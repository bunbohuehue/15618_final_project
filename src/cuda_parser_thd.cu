#include "hip/hip_runtime.h"
#define NUM_THREADS 1024
#define TOLERANCE 0.001
#include <iostream>
#include <tuple>
#include <vector>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cuda_parser.h"
using namespace std;

/* Below are all functions used in sequential version of the parser
	 These will also be useful when implementing threadBasedBinaryRelax
 */

Scores initScores (int nWords, int num_symbol) {
	Scores scores(nWords + 1, vector<vector<float>>(nWords + 1, vector<float>(num_symbol, -FLT_MAX)));
	return scores;
}

void lexiconScores (Scores& scores, vector<string> sen, int nWords, unordered_map<string,
					vector<tuple<string, vector<float>>>> lex, SymToIdx sti, IdxToSym its, Occured& occured) {
	for(int start = 0; start < nWords; start++) {
		string word = sen[start];
		vector<tuple<string, vector<float>>> rules = lex[word];
		for(int i = 0; i < rules.size(); i++) {
			// Extract information from grammar rules
			tuple<string, vector<float>> pair = rules[i];
			string tag = get<0>(pair);
			vector<float> probs = get<1>(pair);
			for(int j = 0; j < probs.size(); j++) {
				string subtag = tag + '_' + to_string(j);
				int tagidx = sti[subtag];
				scores[start][start+1][tagidx] = probs[j];
				occured[0][tagidx] = 1;
			}
		}
	}
}

void binaryRelax (Scores& scores, int nWords,
					int length, BinaryGrammar gr, Occured& occured) {
	for(int i = 0; i < gr.size(); i++) {
		// Extract information from grammar rules
		tuple<int, int, int> pair = get<0>(gr[i]);
		int symbol = get<0>(pair);
		int lsym = get<1>(pair);
		int rsym = get<2>(pair);
		float rulescore = get<1>(gr[i]);
		for(int split = 1; split < length; split++) {
			if (occured[split-1][lsym] && occured[length-split-1][rsym]){
				for(int start = 0; start <= nWords-length; start++) {
					int end = start + length;
					float lscore = scores[start][start+split][lsym];
					if (lscore > -FLT_MAX) {
						float rscore = scores[start+split][end][rsym];
						if (rscore > -FLT_MAX) {
							float current = scores[start][end][symbol];
							float total = rulescore + lscore + rscore;
							if (total > current) {
								scores[start][end][symbol] = total;
								occured[length-1][symbol] = 1;
							}
						}
					}
				}
			}
		}
	}
}

void unaryRelax (Scores& scores, int nWords,
				 int length, UnaryGrammar gr, Occured& occured) {
	for(int i = 0; i < gr.size(); i++) {
		// Extract information from grammar rules
		tuple<int, int> pair = get<0>(gr[i]);
		int symbol = get<0>(pair);
		int lsym = get<1>(pair);
		if (occured[length-1][lsym]) {
			float rulescore = get<1>(gr[i]);
			for(int start = 0; start <= nWords-length; start++) {
				int end = start + length;
				float current = scores[start][end][symbol];
				if(scores[start][end][lsym] > -FLT_MAX) {
					float total = rulescore + scores[start][end][lsym];
					if (total > current) {
						scores[start][end][symbol] = total;
						occured[length-1][symbol] = 1;
					}
				}
			}
		}
	}
}

__global__ void findRoot(float* score_arr, int* root_CUDA, int dim1, int dim2, int dim3) {
	float max = -FLT_MAX;
	int current = 0;
	for (int i = 0; i < dim3; i++) {
		if (score_arr[i*dim1*dim2+(dim2-1)*dim1] > max) {
			max = score_arr[i*dim1*dim2+(dim2-1)*dim1];
			current = i;
		}
	}
	root_CUDA[0] = current;
}

__global__ void searchUnaryRules (float* score_arr, int* lsym_CUDA, UG* ug, int ug_size,
	int dim1, int dim2, int dim3, int start, int end, int sym) {
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadId < ug_size){
			int symbol = ug[threadId].A;
			if (symbol == sym) {
				int lsym = ug[threadId].B;
				float rulescore = ug[threadId].score;
				float symbolscore = score_arr[symbol*dim1*dim2 + end*dim1 + start];
				float lsymscore = score_arr[lsym*dim1*dim2 + end*dim1 + start];
				float diff = symbolscore - lsymscore - rulescore;
				if(diff > -TOLERANCE && diff < TOLERANCE){
					lsym_CUDA[0] = lsym;
				}
			}
		}
}

__global__ void searchBinaryRules (float* score_arr, int* children_CUDA, BG* bg, int bg_size,
	int dim1, int dim2, int dim3, int start, int end, int sym, int* flag) {
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadId < bg_size) {
			int symbol = bg[threadId].A;
			if (symbol == sym) {
				int lsym = bg[threadId].B;
				int rsym = bg[threadId].C;
				float rulescore = bg[threadId].score;
				for(int split = start+1; split <= end-1; split++) {
					float symbolscore = score_arr[symbol*dim1*dim2 + end*dim1 + start];
					float lsymscore = score_arr[lsym*dim1*dim2 + split*dim1 + start];
					float rsymscore = score_arr[rsym*dim1*dim2 + end*dim1 + split];
					float diff = symbolscore - lsymscore -  rsymscore - rulescore;
					if(diff > -TOLERANCE && diff < TOLERANCE){
						if (atomicCAS(flag, 0, 1) == 0){
							children_CUDA[0] = lsym;
							children_CUDA[1] = rsym;
							children_CUDA[2] = split;
						}
					}
				}
			}
		}
}


Ptree* CUDAsearch (float* score_arr, int symidx, vector<string> sen, int start, int end,
	BG* bg, int bg_size, UG* ug, int ug_size, int dim1, int dim2, int dim3, SymToIdx sti, IdxToSym its){
	Ptree* root = (Ptree*) malloc(sizeof(Ptree));
	// Find root if at depth 0
	if (symidx == -1) {
		int* root = new int[1];
		root[0] = -1;
		int* root_CUDA;
		hipMalloc((void**)&root_CUDA, sizeof(int));
		hipMemcpy(root_CUDA, root, sizeof(int), hipMemcpyHostToDevice);
		findRoot<<<1,1>>>(score_arr, root_CUDA, dim1, dim2, dim3);
		hipMemcpy(root, root_CUDA, sizeof(int), hipMemcpyDeviceToHost);
		root->symbol = its[root[0]];
		hipFree(root);
		hipFree(root_CUDA);
	} else {
		root->symbol = its[symidx];
	}
	Ptree* curr = root;

	int* lsym = new int[1];
	lsym[0] = -1;
	int* lsym_CUDA;
	hipMalloc((void**)&lsym_CUDA, sizeof(int));
	hipMemcpy(lsym_CUDA, lsym, sizeof(int), hipMemcpyHostToDevice);
	searchUnaryRules<<<((ug_size+NUM_THREADS)/NUM_THREADS), NUM_THREADS>>>(score_arr, lsym_CUDA, ug, ug_size,
		dim1, dim2, dim3, start, end, sti[curr->symbol]);
	hipMemcpy(lsym, lsym_CUDA, sizeof(int), hipMemcpyDeviceToHost);
	if (lsym[0] != -1) {
		Ptree* child = (Ptree*) malloc(sizeof(Ptree));
		child->symbol = its[lsym[0]];
		root->left = child;
		curr = child;
	}
	hipFree(lsym);
	hipFree(lsym_CUDA);

	if (start+1 == end) {
		Ptree* leaf = (Ptree*) malloc(sizeof(Ptree));
		leaf->symbol = sen[start];
		curr->left = leaf;
		return root;
	}

	int* children = new int[3];
	children[0] = -1;
	children[1] = -1
	children[2] = -1;
	int* children_CUDA;
	hipMalloc((void**)&children_CUDA, 3 * sizeof(int));
	hipMemcpy(children_CUDA, children, 3 * sizeof(int), hipMemcpyHostToDevice);
	int* modified = new int[1];
	modified[0] = 0;
	int* modified_CUDA;
	hipMalloc((void**)&modified_CUDA, sizeof(int));
	hipMemcpy(modified_CUDA, modified, sizeof(int), hipMemcpyHostToDevice);
	searchBinaryRules<<<((bg_size+NUM_THREADS)/NUM_THREADS), NUM_THREADS>>>(score_arr, children_CUDA, bg, bg_size,
		dim1, dim2, dim3, start, end, sti[curr->symbol], modified_CUDA);
	hipMemcpy(children, children_CUDA, 3 * sizeof(int), hipMemcpyDeviceToHost);
	if (children[0] != -1 && children[1] != -1 && children[2] != -1) {
		curr->left = searchHighest(score_arr, children[0], sen, start, children[2],
			bg, bg_size, ug, ug_size, dim1, dim2, dim3, sti, its);
		curr->right = searchHighest(score_arr, children[1], sen, children[2], end,
			bg, bg_size, ug, ug_size, dim1, dim2, dim3, sti, its);
	}
	hipFree(children);
	hipFree(children_CUDA);
	hipFree(modified);
	hipFree(modified_CUDA);
	return root;
}

// Ptree* searchHighest (Scores& scores, int symidx, vector<string> sen,
// 						int start, int end, BinaryGrammar gr2, UnaryGrammar gr1, SymToIdx sti, IdxToSym its){
// 	Ptree* root = (Ptree*) malloc(sizeof(Ptree));
// 	if (symidx == -1) {
// 		float max = -FLT_MAX;
// 		int current = 0;
// 		for (int i = 0; i < scores[0][end].size(); i++) {
// 			if (scores[0][end][i] > max) {
// 				max = scores[0][end][i];
// 				current = i;
// 			}
// 		}
// 		root->symbol = its[current];
// 	} else {
// 		root->symbol = its[symidx];
// 	}
// 	Ptree* curr = root;
	// for(int i = 0; i < gr1.size(); i++) {
	// 	tuple<int, int> pair = get<0>(gr1[i]);
	// 	int symbol = get<0>(pair);
	// 	if (symbol == sti[curr->symbol]) {
	// 		int lsym = get<1>(pair);
	// 		float prob = get<1>(gr1[i]);
	// 		float diff = scores[start][end][symbol] - scores[start][end][lsym] - prob;
	// 		if(diff > -TOLERANCE && diff < TOLERANCE){
	// 			Ptree* child = (Ptree*) malloc(sizeof(Ptree));
	// 			child->symbol = its[lsym];
	// 			root->left = child;
	// 			curr = child;
	// 		}
	// 	}
	// }
// 	if (start+1 == end) {
// 		Ptree* leaf = (Ptree*) malloc(sizeof(Ptree));
// 		leaf->symbol = sen[start];
// 		curr->left = leaf;
// 		return root;
// 	}
// 	for(int j = 0; j < gr2.size(); j++) {
// 		tuple<int, int, int> pair = get<0>(gr2[j]);
// 		int symbol = get<0>(pair);
// 		if (symbol == sti[curr->symbol]) {
// 			int lsym = get<1>(pair);
// 			int rsym = get<2>(pair);
// 			float rscore= get<1>(gr2[j]);
// 			for(int split = start+1; split <= end-1; split++) {
// 				float diff = scores[start][end][symbol] - scores[start][split][lsym] - scores[split][end][rsym] - rscore;
// 				if(diff > -TOLERANCE && diff < TOLERANCE){
// 					curr->left = searchHighest(scores, lsym, sen, start, split, gr2, gr1, sti, its);
// 					curr->right = searchHighest(scores, rsym, sen, split, end, gr2, gr1, sti, its);
// 				}
// 			}
// 		}
// 	}
// 	return root;
// }

float* moveScoreToCUDA(Scores s, int dim1, int dim2, int dim3) {
	float* cuScore;
	int total = dim1*dim2*dim3;
	hipHostMalloc((void**)&cuScore, total * sizeof(float));
	for (int i = 0; i < dim1; i++){
		for (int j = 0; j < dim2; j++) {
			for (int k = 0; k < dim3; k++) {
				cuScore[k*dim1*dim2 + j*dim1 + i] = s[i][j][k];
			}
		}
	}
	float* deviceScore;
	hipMalloc((void**)&deviceScore, total * sizeof(float));
	hipMemcpy(deviceScore, cuScore, total * sizeof(float), hipMemcpyHostToDevice);
	hipHostFree(cuScore);
	return deviceScore;
}

BG* moveBgToCUDA(BinaryGrammar bg) {
	BG* grammars;
	hipHostMalloc((void**)&grammars, bg.size() * sizeof(BG));
	for (int i = 0; i < bg.size(); i++) {
		tuple<int, int, int> pair = get<0>(bg[i]);
		int symbol = get<0>(pair);
		int lsym = get<1>(pair);
		int rsym = get<2>(pair);
		float rulescore = get<1>(bg[i]);
		grammars[i].A = symbol;
		grammars[i].B = lsym;
		grammars[i].C = rsym;
		grammars[i].score = rulescore;
	}
	// allocate CUDA memory
	BG* gr;
	hipMalloc((void**)&gr, bg.size() * sizeof(BG));
	hipMemcpy(gr, grammars, bg.size() * sizeof(BG), hipMemcpyHostToDevice);
	hipHostFree(grammars);
	return gr;
}

UG* moveUgToCUDA(UnaryGrammar ug) {
	UG* grammars;
	hipHostMalloc((void**)&grammars, ug.size() * sizeof(UG));
	for (int i = 0; i < ug.size(); i++) {
		tuple<int, int> pair = get<0>(ug[i]);
		int symbol = get<0>(pair);
		int target = get<1>(pair);
		float rulescore = get<1>(ug[i]);
		grammars[i].A = symbol;
		grammars[i].B = target;
		grammars[i].score = rulescore;
	}
	// allocate CUDA memory
	UG* gr;
	hipMalloc((void**)&gr, ug.size() * sizeof(UG));
	hipMemcpy(gr, grammars, ug.size() * sizeof(UG), hipMemcpyHostToDevice);
	hipHostFree(grammars);
	return gr;
}

__device__ __forceinline__ float atomicMaxFloat (float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));
	return old;
}

__global__ static void UnaryRelaxKernel(UG* ug, float* deviceScores, float* shared_max,
	int rulesize, int dim1, int dim2, int dim3, int num_starts, int spanlen) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < rulesize){
		int symbol = ug[threadId].A;
		int lsym = ug[threadId].B;
		float rulescore = ug[threadId].score;
		for (int start = 0; start < num_starts; start++) {
			float localMax = -FLT_MAX;
			int end = start + spanlen;
			// score_arr is on CUDA
			// [start][split][lsym]
			float lscore;
			lscore = deviceScores[lsym*dim1*dim2 + end*dim1 + start];
			if (lscore > -FLT_MAX) {
				float total = rulescore + lscore;
				if (total > localMax) {
					localMax = total;
				}
			}
			atomicMaxFloat(&shared_max[start*dim3+symbol], localMax);
		}
	}
}

__global__ static void BinaryRelaxKernel(BG* bg, float* deviceScores, float* shared_max,
	int rulesize, int dim1, int dim2, int dim3, int num_starts, int spanlen) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < rulesize){
		int symbol = bg[threadId].A;
		int lsym = bg[threadId].B;
		int rsym = bg[threadId].C;
		float rulescore = bg[threadId].score;
		for (int start = 0; start < num_starts; start++) {
			float localMax = -FLT_MAX;
			int end = start + spanlen;
			// score_arr is on CUDA
			for (int split = start + 1; split <= end - 1; split++) {
				// [start][split][lsym]
				float lscore;
				lscore = deviceScores[lsym*dim1*dim2 + split*dim1 + start];
				// [split][end][rsym]
				float rscore;
				rscore = deviceScores[rsym*dim1*dim2 + end*dim1 + split];
				if (lscore > -FLT_MAX && rscore > -FLT_MAX) {
					float total;
					total = rulescore + lscore + rscore;
					if (total > localMax) {
						localMax = total;
					}
				}
			}
			atomicMaxFloat(&shared_max[start*dim3+symbol], localMax);
		}
	}
}

__global__ void update_score(float* score, float*shared_max, int dim1, int dim2, int num_symbol, int num_starts, int spanlen) {
	for (int i = 0; i < num_starts; i++) {
		int end = i + spanlen;
		for (int j = 0; j < num_symbol; j++) {
			if (score[j*dim1*dim2 + end*dim1 + i] < shared_max[i*num_symbol+j]){
				score[j*dim1*dim2 + end*dim1 + i] = shared_max[i*num_symbol+j];
			}
		}
	}
}

// void RuleBasedUnaryRelax (float* score_arr, int nWords, int length, UG* ug, int ug_size, int num_symbol) {
// 	int dim1 = nWords + 1;
// 	int dim2 = nWords + 1;
// 	for (int start = 0; start <= nWords - length; start++) {
// 		int end = start + length;
//     // score_arr is on CUDA
//     float* shared_max = new float[num_symbol];
// 		for (int i = 0; i < num_symbol; i++) {
//       shared_max[i] = -FLT_MAX;
// 		}
//     float* shared_max_CUDA;
//     hipMalloc((void**)&shared_max_CUDA, num_symbol * sizeof(float));
// 		hipMemcpy(shared_max_CUDA, shared_max, num_symbol * sizeof(float), hipMemcpyHostToDevice);
// 		hipFree(shared_max);
//
// 		UnaryRelaxKernel<<<(ug_size+NUM_THREADS)/NUM_THREADS, NUM_THREADS>>>(ug, score_arr, shared_max_CUDA, ug_size,
// 			nWords+1, nWords+1, num_symbol, start, end);
//
//     // copy back the shared_max array modified by kernel
//     // hipMemcpy(shared_max, shared_max_CUDA, num_symbol * sizeof(float), hipMemcpyDeviceToHost);
//     // update score array
// 		update_score<<<1,1>>>(score_arr, shared_max_CUDA, start, end, dim1, dim2, num_symbol);
// 		hipFree(shared_max_CUDA);
//   }
// }

void RuleBasedUnaryRelax (float* score_arr, int nWords, int length, UG* ug, int ug_size, int num_symbol) {
	// Note that bg is already on device
	// begin of loop body
	// TODO: move this for loop to kernel!!!!!!!!!!
	int dim1 = nWords + 1;
	int dim2 = nWords + 1;

	int num_starts = nWords - length + 1;
	int num_ele = num_symbol * num_starts;
	float* shared_max = new float[num_ele];
	for (int i = 0; i < num_ele; i++) {
		shared_max[i] = -FLT_MAX;
	}
	float* shared_max_CUDA;
	hipMalloc((void**)&shared_max_CUDA, num_ele * sizeof(float));
	hipMemcpy(shared_max_CUDA, shared_max, num_ele * sizeof(float), hipMemcpyHostToDevice);
	hipFree(shared_max);

	UnaryRelaxKernel<<<((ug_size+NUM_THREADS)/NUM_THREADS), NUM_THREADS>>>(ug, score_arr, shared_max_CUDA, ug_size,
		nWords+1, nWords+1, num_symbol, num_starts, length);

	// copy back the shared_max array modified by kernel
	// hipMemcpy(shared_max, shared_max_CUDA, num_symbol * sizeof(float), hipMemcpyDeviceToHost);
	// update score array
	update_score<<<1,1>>>(score_arr, shared_max_CUDA, dim1, dim2, num_symbol, num_starts, length);
	hipFree(shared_max_CUDA);
}

void RuleBasedBinaryRelax (float* score_arr, int nWords, int length, BG* bg, int bg_size, int num_symbol) {
	// Note that bg is already on device
	// begin of loop body
	// TODO: move this for loop to kernel!!!!!!!!!!
	int dim1 = nWords + 1;
	int dim2 = nWords + 1;

	int num_starts = nWords - length + 1;
	int num_ele = num_symbol * num_starts;
	float* shared_max = new float[num_ele];
	for (int i = 0; i < num_ele; i++) {
		shared_max[i] = -FLT_MAX;
	}
	float* shared_max_CUDA;
	hipMalloc((void**)&shared_max_CUDA, num_ele * sizeof(float));
	hipMemcpy(shared_max_CUDA, shared_max, num_ele * sizeof(float), hipMemcpyHostToDevice);
	hipFree(shared_max);

	BinaryRelaxKernel<<<((bg_size+NUM_THREADS)/NUM_THREADS), NUM_THREADS>>>(bg, score_arr, shared_max_CUDA, bg_size,
		nWords+1, nWords+1, num_symbol, num_starts, length);

	// copy back the shared_max array modified by kernel
	// hipMemcpy(shared_max, shared_max_CUDA, num_symbol * sizeof(float), hipMemcpyDeviceToHost);
	// update score array
	update_score<<<1,1>>>(score_arr, shared_max_CUDA, dim1, dim2, num_symbol, num_starts, length);
	hipFree(shared_max_CUDA);
}

// void RuleBasedBinaryRelax (float* score_arr, int nWords, int length, BG* bg, int bg_size, int num_symbol) {
//   // Note that bg is already on device
//   // begin of loop body
// 	// TODO: move this for loop to kernel!!!!!!!!!!
// 	int dim1 = nWords + 1;
// 	int dim2 = nWords + 1;
//   for (int start = 0; start <= nWords - length; start++) {
// 		int end = start + length;
//     // score_arr is already CUDA
// 		float* shared_max = new float[num_symbol];
// 		for (int i = 0; i < num_symbol; i++) {
//       shared_max[i] = -FLT_MAX;
// 		}
//     float* shared_max_CUDA;
//     hipMalloc((void**)&shared_max_CUDA, num_symbol * sizeof(float));
// 		hipMemcpy(shared_max_CUDA, shared_max, num_symbol * sizeof(float), hipMemcpyHostToDevice);
// 		hipFree(shared_max);
//
// 		BinaryRelaxKernel<<<(bg_size+NUM_THREADS)/NUM_THREADS, NUM_THREADS>>>(bg, score_arr, shared_max_CUDA, bg_size,
// 			nWords+1, nWords+1, num_symbol, start, end);
//
//     // copy back the shared_max array modified by kernel
//     // hipMemcpy(shared_max, shared_max_CUDA, num_symbol * sizeof(float), hipMemcpyDeviceToHost);
//     // update score array
// 		update_score<<<1,1>>>(score_arr, shared_max_CUDA, start, end, dim1, dim2, num_symbol);
// 		hipFree(shared_max_CUDA);
//   }
// }

Ptree* parse_sequential(vector<string> sen, unordered_map<string, vector<tuple<string, vector<float>>>> lex,
			 BinaryGrammar bg, UnaryGrammar ug, int num_symbol, SymToIdx sti, IdxToSym its) {
	int nWords = (int)sen.size();
	Scores scores = initScores(nWords, num_symbol);
	Occured occured(nWords, vector<bool>(num_symbol, 0));
	lexiconScores(scores, sen, nWords, lex, sti, its, occured);
	for(int spanlen = 2; spanlen <= nWords; spanlen++) {
		binaryRelax(scores, nWords, spanlen, bg, occured);
		unaryRelax(scores, nWords, spanlen, ug, occured);
	}
	Ptree* result = searchHighest(scores, -1, sen, 0, nWords, bg, ug, sti, its);
	return result;
}

Ptree* RuleBasedParse(vector<string> sen, unordered_map<string, vector<tuple<string, vector<float>>>> lex,
			 BG* gr1, UG* gr2, int num_symbol, SymToIdx sti, IdxToSym its, int bg_size, int ug_size,
		   BinaryGrammar bg, UnaryGrammar ug) {

	int nWords = (int)sen.size();
	Scores scores = initScores(nWords, num_symbol);
	Occured occured(nWords, vector<bool>(num_symbol, 0));
	lexiconScores(scores, sen, nWords, lex, sti, its, occured);

	float* score_arr = moveScoreToCUDA(scores, nWords + 1, nWords + 1, num_symbol);

	for(int spanlen = 2; spanlen <= nWords; spanlen++) {
		RuleBasedBinaryRelax(score_arr, nWords, spanlen, gr1, bg_size, num_symbol);
		RuleBasedUnaryRelax(score_arr, nWords, spanlen, gr2, ug_size, num_symbol);
	}

	int dim1 = nWords + 1;
	int dim2 = nWords + 1;
	int dim3 = num_symbol;
	int total = dim1*dim2*dim3;
	float* hostScore;
	hipHostMalloc((void**)&hostScore, total * sizeof(float));
	hipMemcpy(hostScore, score_arr, total * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < dim1; i++){
		for (int j = 0; j < dim2; j++) {
			for (int k = 0; k < dim3; k++) {
				scores[i][j][k] = hostScore[k*dim1*dim2 + j*dim1 + i];
			}
		}
	}
	hipFree(hostScore);

	Ptree* result = CUDAsearch(score_arr, -1, sen, 0, nWords, gr2, bg_size, gr1, ug_size, dim1,dim2,dim3, sti, its);
	// Ptree* result = searchHighest(scores, -1, sen, 0, nWords, bg, ug, sti, its);
	cout << result->symbol << endl;
	hipFree(score_arr);
	return result;
}

Ptree* BlockBasedParse(vector<string> sen, unordered_map<string, vector<tuple<string, vector<float>>>> lex,
			 BG* gr1, UG* gr2, int num_symbol, SymToIdx sti, IdxToSym its, int bg_size, int ug_size,
		   BinaryGrammar bg, UnaryGrammar ug) {

	int nWords = (int)sen.size();
	Scores scores = initScores(nWords, num_symbol);
	Occured occured(nWords, vector<bool>(num_symbol, 0));
	lexiconScores(scores, sen, nWords, lex, sti, its, occured);

	float* score_arr = moveScoreToCUDA(scores, nWords + 1, nWords + 1, num_symbol);

	for(int spanlen = 2; spanlen <= nWords; spanlen++) {
		BlockBasedBinaryRelax(score_arr, nWords, spanlen, gr1, bg_size, num_symbol);
		BlockBasedUnaryRelax(score_arr, nWords, spanlen, gr2, ug_size, num_symbol);
	}

	int dim1 = nWords + 1;
	int dim2 = nWords + 1;
	int dim3 = num_symbol;
	int total = dim1*dim2*dim3;
	float* hostScore;
	hipHostMalloc((void**)&hostScore, total * sizeof(float));
	hipMemcpy(hostScore, score_arr, total * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < dim1; i++){
		for (int j = 0; j < dim2; j++) {
			for (int k = 0; k < dim3; k++) {
				scores[i][j][k] = hostScore[k*dim1*dim2 + j*dim1 + i];
			}
		}
	}
	hipFree(hostScore);
	hipFree(score_arr);

	// Ptree* result = searchHighest(scores, -1, sen, 0, nWords, bg, ug, sti, its);
	//cout << result->symbol << endl;
	return NULL;
}

void parseAllRuleBased (vector<vector<string>> sen, unordered_map<string, vector<tuple<string, vector<float>>>> lex,
		 BinaryGrammar bg, UnaryGrammar ug, int num_symbol, SymToIdx sti, IdxToSym its, int num_sen) {
	int total = 0;
	int num = 0;
	int bg_size = bg.size();
	int ug_size = ug.size();
	BG* gr1 = moveBgToCUDA(bg);
	UG* gr2 = moveUgToCUDA(ug);
	for (int i = 0; i < num_sen; i++){
		int len = (int)sen[i].size();
		num += 1;
		total += len;
		RuleBasedParse(sen[i], lex, gr1, gr2, num_symbol, sti, its, bg_size, ug_size, bg, ug);
		cout << "Finished parsing sentence (CUDA) " << num << endl;
	}
	hipFree(gr1);
	hipFree(gr2);
	std::cout << "avg len: " << total/num_sen << " \n";
}

void parseAllBlockBased (vector<vector<string>> sen, unordered_map<string, vector<tuple<string, vector<float>>>> lex,
		 BinaryGrammar bg, UnaryGrammar ug, int num_symbol, SymToIdx sti, IdxToSym its, int num_sen) {
	int total = 0;
	int num = 0;
	int bg_size = bg.size();
	int ug_size = ug.size();
	BG* gr1 = moveBgToCUDA(bg);
	UG* gr2 = moveUgToCUDA(ug);
	for (int i = 0; i < num_sen; i++){
		int len = (int)sen[i].size();
		num += 1;
		total += len;
		BlockBasedParse(sen[i], lex, gr1, gr2, num_symbol, sti, its, bg_size, ug_size, bg, ug);
		cout << "Finished parsing sentence (CUDA) " << num << endl;
	}
	hipFree(gr1);
	hipFree(gr2);
	std::cout << "avg len: " << total/num_sen << " \n";
}
